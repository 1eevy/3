#include "hip/hip_runtime.h"
#include <stdint.h>
#include "stencil.h"
#include "float3.h"

// Exchange + Dzyaloshinskii-Moriya interaction according to
// Bagdanov and Röβler, PRL 87, 3, 2001. eq.8 (out-of-plane symmetry breaking).
// Taking into account proper boundary conditions.
// m: normalized magnetization
// H: effective field in Tesla
// D: dmi strength / Msat, in Tesla*m
// A: Aex/Msat
extern "C" __global__ void
adddmi(float* __restrict__ Hx, float* __restrict__ Hy, float* __restrict__ Hz,
       float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
       float Dx, float Dy, float Dz, float A,
       float cx, float cy, float cz, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    int I = idx(ix, iy, iz);                     // central cell index
    float3 h = make_float3(Hx[I], Hy[I], Hz[I]); // add to H
    float3 m = make_float3(mx[I], my[I], mz[I]); // central m
    int i_;

    // x derivatives (along length)
    {
        float3 m1 = make_float3(0.0f, 0.0f, 0.0f); // left neighbor
        float3 m2 = make_float3(0.0f, 0.0f, 0.0f); // right neighbor

        // load neighbor m if inside grid, 0 otherwise
        i_ = idx(lclampx(ix-1), iy, iz);
        if (ix-1 >= 0 || PBCx) {
            m1 = make_float3(mx[i_], my[i_], mz[i_]);
        }

        i_ = idx(hclampx(ix+1), iy, iz);
        if (ix+1 < Nx || PBCx) {
            m2 = make_float3(mx[i_], my[i_], mz[i_]);
        }

        // BC's for zero cells (either due to grid or hole boundaries)
        float Dx_2A = (Dx/(2.0f*A));
        if (dot(m1, m1) == 0.0f) {             // left neigbor missing
            m1.x = m.x - (-cx * Dx_2A * m.z);  // extrapolate to left (-cx)
            m1.y = m.y;
            m1.z = m.z + (-cx * Dx_2A * m.x);
        }
        if (dot(m2, m2) == 0.0f) {            // right neigbor missing
            m2.x = m.x - (cx * Dx_2A * m.z);  // extrapolate to right (+cx)
            m2.y = m.y;
            m2.z = m.z + (cx * Dx_2A * m.x);
        }

        h   += (2.0f*A/(cx*cx)) * ((m1 - m) + (m2 - m)); // exchange
        h.x += Dx*(m2.z-m1.z)/cx;
        h.z -= Dx*(m2.x-m1.x)/cx;
        // note: actually 2*D * delta / (2*c)
    }

    // y derivatives (along height)
    {
        float3 m1 = make_float3(0.0f, 0.0f, 0.0f);
        float3 m2 = make_float3(0.0f, 0.0f, 0.0f);

        i_ = idx(ix, lclampy(iy-1), iz);
        if (iy-1 >= 0 || PBCy) {
            m1 = make_float3(mx[i_], my[i_], mz[i_]);
        }

        i_ = idx(ix, hclampy(iy+1), iz);
        if  (iy+1 < Ny || PBCy) {
            m2 = make_float3(mx[i_], my[i_], mz[i_]);
        }

        float Dy_2A = (Dy/(2.0f*A));
        if (dot(m1, m1) == 0.0f) {
            m1.x = m.x;
            m1.y = m.y - (-cy * Dy_2A * m.z);
            m1.z = m.z + (-cy * Dy_2A * m.y);
        }
        if (dot(m2, m2) == 0.0f) {
            m2.x = m.x;
            m2.y = m.y - (cy * Dy_2A * m.z);
            m2.z = m.z + (cy * Dy_2A * m.y);
        }

        h   += (2.0f*A/(cy*cy)) * ((m1 - m) + (m2 - m));
        h.y += Dy*(m2.z-m1.z)/cy;
        h.z -= Dy*(m2.y-m1.y)/cy;
    }

    // write back, result is H + Hdmi + Hex
    Hx[I] = h.x;
    Hy[I] = h.y;
    Hz[I] = h.z;
}

// Note on boundary conditions.
//
// We need the derivative and laplacian of m in point A, but e.g. C lies out of the boundaries.
// We use the boundary condition in B (derivative of the magnetization) to extrapolate m to point C:
// 	m_C = m_A + (dm/dx)|_B * cellsize
//
// When point C is inside the boundary, we just use its actual value.
//
// Then we can take the central derivative in A:
// 	(dm/dx)|_A = (m_C - m_D) / (2*cellsize)
// And the laplacian:
// 	lapl(m)|_A = (m_C + m_D - 2*m_A) / (cellsize^2)
//
// All these operations should be second order as they involve only central derivatives.
//
//    ------------------------------------------------------------------ *
//   |                                                   |             C |
//   |                                                   |          **   |
//   |                                                   |        ***    |
//   |                                                   |     ***       |
//   |                                                   |   ***         |
//   |                                                   | ***           |
//   |                                                   B               |
//   |                                               *** |               |
//   |                                            ***    |               |
//   |                                         ****      |               |
//   |                                     ****          |               |
//   |                                  ****             |               |
//   |                              ** A                 |               |
//   |                         *****                     |               |
//   |                   ******                          |               |
//   |          *********                                |               |
//   |D ********                                         |               |
//   |                                                   |               |
//   +----------------+----------------+-----------------+---------------+
//  -1              -0.5               0               0.5               1
//                                 x
