#include "hip/hip_runtime.h"
#include "stencil.h"
#include "float3.h"

// Exchange + Dzyaloshinskii-Moriya interaction according to
// Bagdanov and Röβler, PRL 87, 3, 2001. eq.8 (out-of-plane symmetry breaking).
// Taking into account proper boundary conditions.
// m: normalized magnetization
// H: effective field in Tesla
// D: dmi strength / Msat, in Tesla*m
// A: Aex/Msat
extern "C" __global__ void
adddmi(float* __restrict__ Hx, float* __restrict__ Hy, float* __restrict__ Hz,
       float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
       float Dx, float Dy, float Dz, float A,
       float cx, float cy, float cz, int Nx, int Ny, int Nz) {

    int iz = blockIdx.z * blockDim.z + threadIdx.z;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    int I = idx(ix, iy, iz);                     // central cell index
    float3 h = make_float3(Hx[I], Hy[I], Hz[I]); // add to H
    float3 m = make_float3(mx[I], my[I], mz[I]); // central m
    float3 m1, m2;                               // right, left neighbor

    // x derivatives (along length)
    {
        m1 = make_float3(0.0f, 0.0f, 0.0f);
        m2 = make_float3(0.0f, 0.0f, 0.0f);

        // load neighbor m if inside grid, 0 otherwise
        if (ix+1 < Nx) {
            int I1 = idx(ix+1, iy, iz);
            m1 = make_float3(mx[I1], my[I1], mz[I1]);
        }
        if (ix-1 >= 0) {
            int I2 = idx(ix-1, iy, iz);
            m2 = make_float3(mx[I2], my[I2], mz[I2]);
        }

        // BC's for zero cells (either due to grid or hole boundaries)
        float Dx_2A = (Dx/(2.0f*A));
        if (len(m1) == 0.0f) {
            m1.x = m.x - (cx * Dx_2A * m.z);
            m1.y = m.y;
            m1.z = m.z + (cx * Dx_2A * m.x);
        }
        if (len(m2) == 0.0f) {
            m2.x = m.x + (cx * Dx_2A * m.z);
            m2.y = m.y;
            m2.z = m.z - (cx * Dx_2A * m.x);
        }

        h   += (2.0f*A/(cx*cx)) * ((m1 - m) + (m2 - m)); // exchange
        h.z -= Dx*(m1.x-m2.x)/cx;                        // DMI
        h.x += Dx*(m1.z-m2.z)/cx;
        // note: actually 2*D * delta / (2*c)
    }

    // y derivatives (along height)
    {
        m1 = make_float3(0.0f, 0.0f, 0.0f);
        m2 = make_float3(0.0f, 0.0f, 0.0f);

        if (iy+1 < Ny) {
            int I1 = idx(ix, iy+1, iz);
            m1 = make_float3(mx[I1], my[I1], mz[I1]);
        }
        if (iy-1 >= 0) {
            int I2 = idx(ix, iy-1, iz);
            m2 = make_float3(mx[I2], my[I2], mz[I2]);
        }

        float Dy_2A = (Dy/(2.0f*A));
        if (len(m1) == 0.0f) {
            m1.x = m.x;
            m1.y = m.y - (cy * Dy_2A * m.z);
            m1.z = m.z + (cy * Dy_2A * m.y);
        }
        if (len(m2) == 0.0f) {
            m2.x = m.x;
            m2.y = m.y + (cy * Dy_2A * m.z);
            m2.z = m.z - (cy * Dy_2A * m.y);
        }

        h   += (2.0f*A/(cy*cy)) * ((m1 - m) + (m2 - m));
        h.z -= Dy*(m1.y-m2.y)/cy;
        h.y += Dy*(m1.z-m2.z)/cy;
    }

    // write back, result is H + Hdmi + Hex
    Hx[I] = h.x;
    Hy[I] = h.y;
    Hz[I] = h.z;
}

// Note on boundary conditions.
//
// We need the derivative and laplacian of m in point A, but e.g. C lies out of the boundaries.
// We use the boundary condition in B (derivative of the magnetization) to extrapolate m to point C:
// 	m_C = m_A + (dm/dx)|_B * cellsize
//
// When point C is inside the boundary, we just use its actual value.
//
// Then we can take the central derivative in A:
// 	(dm/dx)|_A = (m_C - m_D) / (2*cellsize)
// And the laplacian:
// 	lapl(m)|_A = (m_C + m_D - 2*m_A) / (cellsize^2)
//
// All these operations should be second order as they involve only central derivatives.
//
//    ------------------------------------------------------------------ *
//   |                                                   |             C |
//   |                                                   |          **   |
//   |                                                   |        ***    |
//   |                                                   |     ***       |
//   |                                                   |   ***         |
//   |                                                   | ***           |
//   |                                                   B               |
//   |                                               *** |               |
//   |                                            ***    |               |
//   |                                         ****      |               |
//   |                                     ****          |               |
//   |                                  ****             |               |
//   |                              ** A                 |               |
//   |                         *****                     |               |
//   |                   ******                          |               |
//   |          *********                                |               |
//   |D ********                                         |               |
//   |                                                   |               |
//   +----------------+----------------+-----------------+---------------+
//  -1              -0.5               0               0.5               1
//                                 x
