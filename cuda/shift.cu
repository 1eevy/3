#include "hip/hip_runtime.h"
#include "stencil.h"

// Copies src to dst, shifting elements by (u, v, w).
// Clamps at the boundaries.
extern "C" __global__ void
shift(float* __restrict__  dst, float* __restrict__  src,
      int Nx,  int Ny,  int Nz, int shx, int shy, int shz) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix < Nx && iy < Ny && iz < Nz) {
        dst[idx(ix, iy, iz)] = src[idxclamp(ix-shx, iy-shy, iz-shz)];
    }
}

