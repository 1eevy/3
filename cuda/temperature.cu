
#include <hip/hip_runtime.h>

extern "C" __global__ void
addtemperature(float* __restrict__  B,      float* __restrict__ noise, float kB2_VgammaDt,
               float* __restrict__ tempRedLUT, int8_t* __restrict__ regions, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        int8_t reg  = regions[i];
        float alphaT_Bs  = tempRedLUT[reg];
        B[i] += noise[i] * sqrtf( kB2_VgammaDt * alphaT_Bs );
    }
}

