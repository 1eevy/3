
#include <hip/hip_runtime.h>
// 2D Z (out-of-plane only) micromagnetic kernel multiplication:
// Mz = Kzz * Mz
//
// ~kernel has mirror symmetry along Y-axis,
// apart form first row,
// and is only stored (roughly) half:
//
// K00:
// xxxxx
// aaaaa
// bbbbb
// ....
// bbbbb
// aaaaa
//
extern "C" __global__ void
kernmulRSymm2Dz(float* __restrict__  fftMz, float* __restrict__  fftKzz, int Nx, int Ny) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if(ix>= Nx || iy>=Ny) {
        return;
    }

    int I = iy*Nx + ix; // linear index for upper half of kernel
    int e = 2 * I;

    float reMz = fftMz[e  ];
    float imMz = fftMz[e+1];

    float Kzz;
    if (iy < Ny/2 + 1) {
        Kzz = fftKzz[I];
    } else {
        int I2 = (Ny-iy)*Nx + ix; // linear index for re-use of lower half
        Kzz = fftKzz[I2];
    }

    fftMz[e  ] = reMz * Kzz;
    fftMz[e+1] = imMz * Kzz;
}

