#include "hip/hip_runtime.h"

#include "float3.h"

// dst = v * dot(a,b)
extern "C" __global__ void
dotproduct(float* __restrict__ dst,
           float* __restrict__ ax, float* __restrict__ ay, float* __restrict__ az,
           float* __restrict__ bx, float* __restrict__ by, float* __restrict__ bz,
           float* __restrict__ vol, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        float v = (vol == NULL? 1.0f: vol[i]);
        float3 A = {ax[i], ay[i], az[i]};
        float3 B = {bx[i], by[i], bz[i]};
        dst[i] = v * dot(A, B);
    }
}

