
#include <hip/hip_runtime.h>
// 2D XY (in-plane) micromagnetic kernel multiplication:
// |Mx| = |Kxx Kxy| * |Mx|
// |My|   |Kyx Kyy|   |My|
//
// ~kernel has mirror symmetry along Y-axis,
// apart form first row,
// and is only stored (roughly) half:
//
// K11, K22:
// xxxxx
// aaaaa
// bbbbb
// ....
// bbbbb
// aaaaa
//
// K12:
// xxxxx
// aaaaa
// bbbbb
// ...
// -aaaa
// -bbbb
extern "C" __global__ void
kernmulRSymm2Dxy(float* __restrict__  fftMx,  float* __restrict__  fftMy,
                 float* __restrict__  fftKxx, float* __restrict__  fftKyy, float* __restrict__  fftKxy,
                 int Nx, int Ny) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if(ix>= Nx || iy>=Ny) {
        return;
    }


    int I = iy*Nx + ix;       // linear index for upper half of kernel
    int e = 2 * I;

    float reMx = fftMx[e  ];
    float imMx = fftMx[e+1];
    float reMy = fftMy[e  ];
    float imMy = fftMy[e+1];

    float Kyy, Kxx, Kxy;
    if (iy < Ny/2 + 1) {
        Kxx = fftKxx[I];
        Kyy = fftKyy[I];
        Kxy = fftKxy[I];
    } else {
        int I2 = (Ny-iy)*Nx + ix; // linear index for re-use of lower half
        Kxx =  fftKxx[I2];
        Kyy =  fftKyy[I2];
        Kxy = -fftKxy[I2]; // !
    }

    fftMx[e  ] = reMx * Kxx + reMy * Kxy;
    fftMx[e+1] = imMx * Kxx + imMy * Kxy;
    fftMy[e  ] = reMx * Kxy + reMy * Kyy;
    fftMy[e+1] = imMx * Kxy + imMy * Kyy;
}

