#include "hip/hip_runtime.h"
#include "stencil.h"

// Copy src (size S, smaller) into dst (size D, larger),
// and multiply by Bsat as defined in regions.
extern "C" __global__ void
copypadmul(float* __restrict__ dst, int Dx, int Dy, int Dz,
           float* __restrict__ src, float* __restrict__ vol, int Sx, int Sy, int Sz,
           float* __restrict__ BsatLUT, int8_t* __restrict__ regions) {

    int iz = blockIdx.z * blockDim.z + threadIdx.z;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (iz<Sz && iy<Sy && ix<Sx) {
        int sI = index(ix, iy, iz, Sx, Sy, Sz); //(iz*Sy + iy)*Sx + k; // source index
        float Bsat = BsatLUT[regions[sI]];
        float v = (vol == NULL? 1.0f: vol[sI]);
        dst[index(ix, iy, iz, Dx, Dy, Dz)] = Bsat * v * src[sI];
    }
}

